#include <stdio.h>

class Managed
{
    public:
        void *operator new(size_t len) {
            void *ptr;
            hipMallocManaged(&ptr, len);
            return ptr;
        }

        void operator delete(void *ptr) {
            hipFree(ptr);
        }
};

template<typename T>
struct Collection
{
    T* elements;
    int count;
};

tempalte<typename T>
T* managedArray(int size)
{
    void *ptr;
    hipMallocManaged(& ptr, size*sizeof(T));
    return (T*)ptr;
}





void main()
{
    printf("%d\n", (3 * 5 * 1 * 2 * 4));

}